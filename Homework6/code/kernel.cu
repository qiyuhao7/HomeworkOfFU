#include "hip/hip_runtime.h"
#include <math.h>
//CUDA RunTime API
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define BLOCK_SIZE 16


__global__ static void matMultCUDA(const float *A, const float* B, float *C, int m, int n, int e)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x *blockDim.x + threadIdx.x;
	if (row < m && col < n)
	{
		float Cvalue = 0;
		for (int i = 0; i < e; i++)
		{
			Cvalue += A[row*e + i] * B[i*n + col];
		}
		C[n*row + col] = Cvalue;
	}


}


extern "C"
void mulWithCUDA(const float *A, const float *B, float *C, int m, int n, int e)
{
	hipSetDevice(0);

	int heightA = m;
	int widthA = e;
	int  heightB = e;
	int widthB = n;
	int heightC = heightA;
	int widthC = widthB;

	float *d_A, *d_B, *d_C;

	hipMalloc((void**)&d_A, sizeof(float)*heightA*widthA);
	hipMalloc((void**)&d_B, sizeof(float)*heightB*widthB);
	hipMalloc((void**)&d_C, sizeof(float)*heightC*widthC);

	hipMemcpy(d_A, A, sizeof(float)*heightA*widthA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeof(float)*heightB*widthB, hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//int numBlock = (widthB*heightA + BLOCK_SIZE*BLOCK_SIZE -1) / BLOCK_SIZE*BLOCK_SIZE;
	dim3 dimGrid(1, 1);
	dimGrid.x = ceil(float(widthB) / double(dimBlock.x));
	dimGrid.y = ceil(float(heightA) / double(dimBlock.y));
	//dim3 dimGrid((widthB + dimBlock.x - 1) / dimBlock.x, (heightA + dimBlock.y - 1) / dimBlock.y);
	//dim3 dimGrid((widthB ) / dimBlock.x, (heightA ) / dimBlock.y);
	matMultCUDA << <dimGrid, dimBlock >> > (d_A, d_B, d_C, heightC, widthC, widthA);

	hipMemcpy(C, d_C, sizeof(float)*heightC*widthC, hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);


}